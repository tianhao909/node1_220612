#include "hip/hip_runtime.h"
#include "/usr/local/cuda-11.3/targets/x86_64-linux/include/hip/hip_runtime.h"
#include<stdlib.h>
#include <stdio.h>
#include <assert.h>

#define N 10

__global__ void gpu(int num) {
    printf("%d\n",num);
}

int main(){
    for(int i=0; i<N; i++){
        //gpu<<<1,2>>>(i);
        hipStream_t stream;
        hipStreamCreate(&stream);
        gpu<<<1,1,0,stream>>>(i);
        hipStreamDestroy(stream);
    }
    hipDeviceSynchronize();

}
